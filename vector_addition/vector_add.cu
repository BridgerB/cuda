#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 1024
#define THREADS_PER_BLOCK 256

__global__ void vector_add(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    size_t size = N * sizeof(float);

    // Allocate host memory
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c = (float*)malloc(size);

    // Initialize input vectors
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy vectors from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel
    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    vector_add<<<blocks, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);

    // Copy result from device to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Verify result
    bool success = true;
    for (int i = 0; i < N; i++) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            success = false;
            break;
        }
    }

    if (success) {
        printf("Vector addition successful!\n");
        printf("Added %d elements: C[0] = %.1f, C[%d] = %.1f\n",
               N, h_c[0], N-1, h_c[N-1]);
    } else {
        printf("Vector addition failed!\n");
    }

    // Free memory
    free(h_a); free(h_b); free(h_c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}