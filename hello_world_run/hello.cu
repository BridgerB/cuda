#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_kernel() {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello World from thread %d\n", tid);
}

int main() {
    const int num_threads = 256;
    const int threads_per_block = 256;
    const int num_blocks = (num_threads + threads_per_block - 1) / threads_per_block;

    hipError_t err;

    hello_kernel<<<num_blocks, threads_per_block>>>();

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA kernel launch error: %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA synchronization error: %s\n", hipGetErrorString(err));
        return 1;
    }

    return 0;
}